#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>

#define true 1
#define false 0

__global__ void localSearch(int **targetBuffer_d, int **solutionBuffer_d, float *solutionBufferValue_d, int **permutationPool_d,
                            int **permutationPoolInverse_d, float ***W_d, int numBits, int numPermutations, int numSegments)
{
    // Energy difference delta_i declared in shared memory (see definition of delta_i in paper)
    // Array size is static, either set an upper bound, or use extern keyword to dynamically specify size (where there should be modifications on calling the kernel function)
    // We use static array size for now, can change later

    // Record the current solution, best solution, and their corresponding values in shared memory, so that all the threads can see them
    __shared__ float delta[512];
    __shared__ int currentSolution[512] = {0}; // Initialize the current solution to be a vector of zeros
    __shared__ int bestSolution[512] = {0};
    __shared__ float min_delta = FLT_MAX;
    __shared__ float currentValue = 0;
    __shared__ float bestValue = FLT_MAX;

    // newly added for straight search and cyclic-min
    __shared__ int k = 0;
    __shared__ int flag = numBits;
    __shared__ int isInitial = true;
    __shared__ int numSegBits = (numBits - 1) / numSegments + 1;
    // Record target solution we want to approach
    // And do bit permutation, the permutation used is blockIdx.x mod numPermutations
    __shared__ int targetSolution[512];
    int pIndex = blockIdx.x % numPermutations;

    // Permutation
    if (threadIdx.x < numBits)
        targetSolution[permutationPool_d[pIndex][threadIdx.x]] = targetBuffer_d[blockIdx.x][threadIdx.x];
    __syncthreads();
    // REMARK: the weight matrix used here is W_d[pIndex] (it is a pointer to pointer of float)
    // One question needs discussion: should we copy the permutated weight matrix on shared memory of this block? The matrix is large

    // Implement straight search (Exactly the same as Algorithm 3 in paper)
    while (flag != 0) // flag is set to be numBits, when every elem of array does not change, we get T from 0.
    {
        flag = numBits; // reset flag
        min_delta = FLT_MAX;

        __syncthreads();

        if (currentSolution[threadIdx.x] != targetSolution[threadIdx.x]) // only calculate when the bit is different from target
        {
            if (isInitial == true) // Initialization from vector 0; using formula in the literature to compute
            {
                delta[threadIdx.x] = W_d[pIndex][threadIdx.x][threadIdx.x];
                for (int j = 0; j < numBits; j++)
                    delta[threadIdx.x] += 2 * W_d[pIndex][threadIdx.x][j] * currentSolution[j];
                delta[threadIdx.x] *= -2 * currentSolution[threadIdx.x] + 1; // \phi(x)
            }
            __syncthreads();
            isInitial = false;

            if (delta[threadIdx.x] < min_delta)
            {
                min_delta = delta[threadIdx.x];
                k = threadIdx.x;
            }
        }

        __syncthreads();

        // update each delta i
        if (thread.Idx == k) // we only need to update once in each block, rather than for all threads
        {
            currentValue += min_delta;
            if (currentValue < bestValue)
            {
                bestValue = currentValue;
                for (int i = 0; i < numBits; i++) // Since in one thread we cannot use parallel computing, we have to use loop
                    bestSolution[i] = currentSolution[i];
            }
            delta[threadIdx.x] *= -1;
        }
        else
        {
            delta[threadIdx.x] += 2 * W_d[pIndex][threadIdx.x][k] * (-2 * currentSolution[threadIdx.x] + 1) * (-2 * currentSolution[k] + 1);
        }

        // flip with minimum delta k
        if (threadIdx.x == 0)
            currentSolution[k] = 1 - currentSolution[k];

        __syncthreads();

        // judge whether to get out of while-loop
        if (currentSolution[threadIdx.x] == targetSolution[threadIdx.x])
            flag--;

        __syncthreads();
    }

    // Now that we know energy value of the target solution,
    // Implement cyclic-Min
    // Question: each segment is dealt with sequentially, where no parallelism exists between segments, wasting other threads
    for (int i = 0; i < numBits; i += numSegBits)
    {
        min_delta = FLT_MAX;
        __syncthreads();

        if (threadIdx.x >= i && threadIdx.x < i + numSegBits)
        {
            delta[threadIdx.x] = W_d[pIndex][threadIdx.x][threadIdx.x];
            for (int j = 0; j < numBits; j++)
                delta[threadIdx.x] += 2 * W_d[pIndex][threadIdx.x][j] * currentSolution[j];
            delta[threadIdx.x] *= -2 * currentSolution[threadIdx.x] + 1;

            if (delta[threadIdx.x] < min_delta)
            {
                min_delta = delta[threadIdx.x];
                k = threadIdx.x;
            }

            __syncthreads();

            if (thread.Idx == k)
                currentValue += min_delta;

            if (currentValue < bestValue)
            {
                bestValue = currentValue;
                if (threadIdx.x < numBits)
                    bestSolution[threadIdx.x] = currentSolution[threadIdx.x];
            }

            __syncthreads();

            if (thread.Idx == k)
                currentSolution[k] = 1 - currentSolution[k];
        }
    }

    // Reverse back the bestSolution, and write it on solution buffer
    if (threadIdx.x < numBits)
    {
        solutionBuffer_d[blockIdx.x][threadIdx.x] = bestSolution[permutationPoolInverse_d[pIndex][threadIdx.x]];
    }
    solutionBufferValue_d[blockIdx.x] = bestValue;
}